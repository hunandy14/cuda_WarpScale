/*****************************************************************
Name : 
Date : 2018/01/08
By   : CharlotteHonG
Final: 2018/01/08
*****************************************************************/
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <vector>
#include <string>
using namespace std;

#include "bilinear.cuh"
#include "OpenBMP.hpp"
#include "CudaMem\CudaMem.cuh"
#include "Timer.hpp"

using uch = unsigned char;

vector<float> tofloat(const uch* img, size_t size) {
	vector<float> temp(size);
	for(size_t i = 0; i < size; i++) {
		temp[i] = img[i];
	} return temp;
}
vector<uch> touch(const float* img, size_t size) {
	vector<uch> temp(size);
	for(size_t i = 0; i < size; i++) {
		temp[i] = img[i];
	} return temp;
}

int main(){
	Timer T;
	// Ū��
	ImgData src("img//kanna.bmp");
	ImgData srcGray, dst, temp;
	T.start();
	srcGray = src.toConvertGray();
	T.print("��Ƕ���");

	// �B�z
	double ratio = 5;
	vector<float> img_gpuRst, img_data = tofloat(srcGray.raw_img.data(), srcGray.size());

	double time;
	//time = biliner_share(img_gpuRst, img_data, srcGray.width, srcGray.height, ratio);
	//time = biliner_CPU(img_gpuRst, img_data, srcGray.width, srcGray.height, ratio);

	WarpScale_rgb(src, temp, 2);
	WarpScale_rgb(temp, dst, 0.5);
	//dst.resize(srcGray);
	//cucopy(srcGray.raw_img, dst.raw_img, srcGray.width, srcGray.height);
	dst.bmp("copyImg.bmp");

	// ��X
	//vector<unsigned char> img_out =  touch(img_gpuRst.data(), img_gpuRst.size());
	//string name = "img//Out-texture_"+to_string(time)+".bmp";
	//string name = "GpuOut.bmp";
	//OpenBMP::bmpWrite(name.c_str(), img_out, srcGray.width*ratio, srcGray.height*ratio, 8);

	return 0;
}